#include "hip/hip_runtime.h"
#include "matrix.h"
#include <stdlib.h>
#include <string.h>

#define MIN(a,b) (((a)<(b))?(a):(b))
#define MAX(a,b) (((a)>(b))?(a):(b))

matrix_t * alloc_matrix(unsigned rows, unsigned columns)
{
    matrix_t * res = (matrix_t*) malloc( sizeof(matrix_t) );
    res->m = (double *) calloc(columns * rows, sizeof(double));
    res->columns = columns;
    res->rows = rows;
    return res;
}

void destroy_matrix(matrix_t *m)
{
    //printf("free %p %p\n", m, m->m);
    free(m->m);
    free(m);
}

void print_matrix(matrix_t *m, bool is_short){
    unsigned lim_rows = 0;
    unsigned lim_col = 0;

    if (is_short)
    {
        lim_rows = MIN(m->rows, 4);
        lim_col = MIN(m->columns, 10);
    }
    else
    {
        lim_rows = m->rows;
        lim_col = m->columns;
    }

    for (int row = 0; row < lim_rows; row ++)
    {
        for (int col = 0; col < lim_col; col ++)
        {
            printf("%.2lf ", m->m[col + row * m->columns]);
        }
        if (is_short && lim_col != m->columns) printf("...");
        printf("\n");
    }
    if (is_short && lim_rows != m->rows) printf("...\n");
}

void hadamard_product(matrix_t *m1, matrix_t *m2, matrix_t *res)
{
    assert ( (m1->columns == m2->columns)   &&
             (m1->columns == res->columns)  &&
             (m1->rows == m2->rows)         &&
             (m1->rows == res->rows));

    for (int idx = 0; idx < m1->rows * m1->columns; idx ++)
    {
            res->m[idx] = m1->m[idx] * m2->m[idx];
    }
}

void matrix_sum(matrix_t *m1, matrix_t *m2, matrix_t *res)
{
    assert ( (m1->columns == m2->columns)  &&
             (m1->columns == res->columns) &&
             (m1->rows == m2->rows)        &&
             (m1->rows == res->rows));

    for (int idx = 0; idx < m1->rows * m1->columns; idx ++)
    { 
        res->m[idx] = m1->m[idx] + m2->m[idx];
    }
}

void matrix_minus(matrix_t *m1, matrix_t *m2, matrix_t *res)
{
    assert ( (m1->columns == m2->columns)  &&
             (m1->columns == res->columns) &&
             (m1->rows == m2->rows)        &&
             (m1->rows == res->rows));
             
    for (int idx = 0; idx < m1->rows * m1->columns; idx ++)
    {
        res->m[idx] = m1->m[idx] - m2->m[idx];
    }
}

void matrix_dot(matrix_t *m1, matrix_t *m2, matrix_t *res)
{
    assert ( (m1->columns == m2->rows)  &&
             (m1->rows == res->rows)    &&
             (m2->columns == res->columns));

    for (int row = 0; row < m1->rows; row ++)
    {
        for (int col = 0; col < m2->columns; col ++)
        {
            int idx = col + row * m2->columns;
            double var = 0.0;

            for (int ii = 0; ii < m1->columns; ii++)
            {
                var += m1->m[ii + row * m1->columns] * m2->m[col + ii * m2->columns];
            }

            res->m[idx] = var;
        }
    }
}




void matrix_function(matrix_t *m1, double (*f)(double), matrix_t *res)
{
    assert ( (m1->columns == res->columns) &&             
             (m1->rows == res->rows));

    for (int idx = 0; idx < m1->rows * m1->columns; idx ++)
    {
        res->m[idx] = f(m1->m[idx]);
    }
}

void matrix_transpose(matrix_t *m1, matrix_t *res)
{
    assert ( (m1->columns == res->rows) &&             
             (m1->rows == res->columns));
    
    for (int row = 0; row < m1->rows; row++)
    {
        for (int col = 0; col < m1->columns; col ++)
        {
            res->m[row + col * m1->rows] = m1->m[col + row * m1->columns];
        }
    }
}

void matrix_scalar(matrix_t *m1, double s, matrix_t *res)
{
    assert ( (m1->rows == res->rows) &&             
             (m1->columns == res->columns));

    for (int idx = 0; idx < m1->columns*m1->rows; idx ++)
    {
        res->m[idx] = m1->m[idx] * s;
    }
}

void matrix_memcpy(matrix_t *dest, const matrix_t *src)
{
    assert ( (dest->rows == src->rows)      &&             
             (dest->columns == src->columns));

    memcpy(dest->m, src->m, src->columns * src->rows * sizeof(double));     
}


__global__ 
void computeMatrixMulGPU
(
   double *A, double *B, double *C,
   int numARows, int numAColumns,
   int numBRows, int numBColumns
)
{
   int row = blockIdx.y * blockDim.y + threadIdx.y;
   int col = blockIdx.x * blockDim.x + threadIdx.x;

   if(row < numARows && col < numBColumns) {
       double sum = 0.0f;
       for (int i = 0; i < numAColumns; ++i) {
           sum += A[row * numAColumns + i] * B[i * numBColumns + col];
       }
       C[row * numBColumns + col] = sum;
   }
}


void matrix_dot_gpu(const matrix_t *m1, const matrix_t *m2, matrix_t *res) {
    // Calculate the size of each matrix in bytes
    

    size_t size_m1 = m1->rows * m1->columns * sizeof(double);
    size_t size_m2 = m2->rows * m2->columns * sizeof(double);
    size_t size_res = res->rows * res->columns * sizeof(double);

   double *deviceA;
   double *deviceB;
   double *deviceC;

   // Memory allocation on the GPU
   hipMalloc((void **)&deviceA, size_m1);
   hipMalloc((void **)&deviceB, size_m2);
   hipMalloc((void **)&deviceC, size_res);
    
    hipMemset(deviceC, 0, size_res); // Initialisez la mémoire à zéro
 
    // Copy matrices from the host to the device
    hipMemcpy(deviceA, m1->m, size_m1, hipMemcpyHostToDevice);
    hipMemcpy(deviceB, m2->m, size_m2, hipMemcpyHostToDevice);
    
 
   dim3 blockDim(16, 16);
   dim3 gridDim(ceil(((double)m2->columns) / blockDim.x), ceil(((double)m1->rows) / blockDim.y));


    computeMatrixMulGPU<<<gridDim, blockDim>>>(deviceA,deviceB, deviceC,m1->rows, m1->columns, m2->rows, m2->columns);
    hipDeviceSynchronize();

    // Copy the result matrix back to the host
    hipMemcpy(res->m, deviceC, size_res, hipMemcpyDeviceToHost);
    
                                                   
    // Free the device matrices
    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);
}



